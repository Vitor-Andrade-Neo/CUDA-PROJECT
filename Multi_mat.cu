#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void MultiMat(float *a, float *b, float *c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        float soma = 0;
        for (int k = 0; k < n; k++) {
            soma += a[i* n + k] * b[k * n + j];
        }
        c[i * n + j] = soma;
    }
}

int main() {
    int n, x, y;
    cin >> n;

    float *a, *b, *c;
    a = (float*)malloc(n * n * sizeof(float));
    b = (float*)malloc(n * n * sizeof(float));
    c = (float*)malloc(n * n * sizeof(float));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            cin >> x >> y;
            a[i * n + j] = x;
            b[i * n + j] = y;
            c[i * n + j] = 0.0f;
        }
    }

    float *da, *db, *dc;
    hipMalloc((void **)&da, n * n * sizeof(float));
    hipMalloc((void **)&db, n * n * sizeof(float));
    hipMalloc((void **)&dc, n * n * sizeof(float));

    hipMemcpy(da, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, n * n * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(32, 16);
    dim3 dimGrid((n + 31) / 32, (n + 15) / 16);
    
    MultiMat<<<dimGrid, dimBlock>>>(da, db, dc, n);
    hipDeviceSynchronize();

    hipMemcpy(c, dc, n * n * sizeof(float), hipMemcpyDeviceToHost);

    printf("C:\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.2f ", c[i * n + j]);
        }
        printf("\n");
    }

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);

    return 0;
}
